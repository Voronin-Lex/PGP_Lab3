#include <hip/hip_runtime.h>


#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define BLOCKSIZE 512


__host__ unsigned int getMax(unsigned int*  Data, int n)
{
   unsigned  int mx = Data[0];
    for (int i = 1; i < n; i++)
        if (Data[i] > mx)
            mx = Data[i];
    return mx;
}

__global__ void FixUpScan(unsigned int* PreScan, unsigned int* PreScannedAuxiliary, unsigned int Size)
{
	int tid = threadIdx.x;
	int BlockOffset=0;

	for(int offset = 2*blockIdx.x*blockDim.x; offset<Size; offset+=2*blockDim.x*gridDim.x)
 {
	 if((BlockOffset*gridDim.x+blockIdx.x)<(Size+2*BLOCKSIZE-1)/(2*BLOCKSIZE))
	{
		if ((offset+2*tid) < Size)
		PreScan[offset+2*tid]+=PreScannedAuxiliary[BlockOffset*gridDim.x+blockIdx.x];

		if ((offset+2*tid+1) < Size)
		PreScan[offset+2*tid+1]+=PreScannedAuxiliary[BlockOffset*gridDim.x+blockIdx.x];
	}

	BlockOffset++;
  }

}

__global__ void GetDiff_N_Sn(unsigned char* Bits, unsigned int* PreScan, unsigned int* Diff_N_Sn, int Size)
{
	*Diff_N_Sn = Size - PreScan[Size-1];
	if(Bits[Size-1]==1) (*Diff_N_Sn)--;
	Bits[Size-1]=Bits[Size-1];   //��� �����������


}
 


__global__ void Sort(unsigned int* InData, unsigned int* PreScan, unsigned int* OutData, unsigned char* Bits, unsigned int* Diff_N_Sn, int Size)
{
	
  
	for(int index = blockIdx.x*blockDim.x+threadIdx.x; index<Size; index+=blockDim.x*gridDim.x)
   {
	if(index<Size)
	{

	if(Bits[index]==0)
	{
		OutData[index - PreScan[index]] = InData[index];
	}

	else
	  OutData[PreScan[index]+(*Diff_N_Sn)] = InData[index];
      		 
     }

   }
}

__global__ void Exchange(unsigned int* InData, unsigned int* OutData, int Size)
{
	for(int index=blockIdx.x*blockDim.x+threadIdx.x; index<Size; index+=blockDim.x*gridDim.x)
	{
		InData[index]=OutData[index];	
	}
}

__global__ void KernelPrescan(unsigned int* Data, unsigned char* Bits, unsigned int* PreScan, unsigned int* Auxiliary, int Size, int bit)   
{
	extern __shared__	unsigned int Tmp[];

	int tid = threadIdx.x;
	int AuxiliaryIndex=0;

	for(int OffsetTid = 2*blockIdx.x*blockDim.x; OffsetTid<Size; OffsetTid+=2*blockDim.x*gridDim.x)
  {
	int offset=1;

	if(OffsetTid+tid<Size)
	 {
		 Tmp[tid] =  (Data[OffsetTid+tid]>>bit)&1;
	     Bits[OffsetTid+tid]=(Data[OffsetTid+tid]>>bit)&1;
	 }
	else
     Tmp[tid] = 0;



	if(OffsetTid+tid+blockDim.x<Size)
	{
		Tmp[tid+blockDim.x] =  (Data[OffsetTid+tid+blockDim.x]>>bit)&1;
		Bits[OffsetTid+tid+blockDim.x] = (Data[OffsetTid+tid+blockDim.x]>>bit)&1;
	}
	 else
     Tmp[tid+blockDim.x] = 0;
	


	for (int d = blockDim.x; d > 0; d>>=1)
	{
		__syncthreads();

		if(tid<d)
		{
	        int ai = offset*(2*tid+1)-1;	
		    int bi = offset*(2*tid+2)-1;	

			Tmp[bi]+=Tmp[ai];
		}
		offset*=2;
	}

	if(tid==0)  
    
	{  if((gridDim.x*AuxiliaryIndex+blockIdx.x)<(Size+2*BLOCKSIZE-1)/(2*BLOCKSIZE))
	    Auxiliary[gridDim.x*AuxiliaryIndex+blockIdx.x]=Tmp[2*blockDim.x-1];
	   
	    Tmp[2*blockDim.x-1]=0; 
	} 

	for(int d=1; d<2*blockDim.x; d*=2)
	{
		offset>>=1;
		__syncthreads();


		if(tid<d)
		{
		   int ai = offset*(2*tid+1)-1;	
		   int bi = offset*(2*tid+2)-1;	
		
		   int t = Tmp[ai];
		   Tmp[ai]=Tmp[bi];
		   Tmp[bi]+=t;
		
		}	
	}

	__syncthreads();


	if((OffsetTid+2*tid)<Size)                         
	PreScan[OffsetTid+2*tid] =  Tmp[2*tid];

	if((OffsetTid+2*tid+1)<Size)
	PreScan[OffsetTid+2*tid+1] = Tmp[2*tid+1]; 

	__syncthreads();
 
	AuxiliaryIndex++;
	}
}


__global__ void KernelPrescanRecursive(unsigned int* PreScan, unsigned int* Auxiliary, int Size, int LastLevel)  
{
	extern __shared__	unsigned int Tmp[];

	int tid = threadIdx.x;
	int AuxiliaryIndex=0;

	for(int OffsetTid = 2*blockIdx.x*blockDim.x; OffsetTid<Size; OffsetTid+=2*blockDim.x*gridDim.x)
  {
	int offset=1;

	if(OffsetTid+tid<Size)
	 {
		 Tmp[tid] =  PreScan[OffsetTid+tid];
	 }
	else
     Tmp[tid] = 0;



	if(OffsetTid+tid+blockDim.x<Size)
	{
		Tmp[tid+blockDim.x] =  PreScan[OffsetTid+tid+blockDim.x];
	}
	 else
     Tmp[tid+blockDim.x] = 0;
	


	for (int d = blockDim.x; d > 0; d>>=1)
	{
		__syncthreads();

		if(tid<d)
		{
	        int ai = offset*(2*tid+1)-1;	
		    int bi = offset*(2*tid+2)-1;	

			Tmp[bi]+=Tmp[ai];
		}
		offset*=2;
	}

	if(tid==0)  
    
	{  if( ((gridDim.x*AuxiliaryIndex+blockIdx.x)<(Size+2*BLOCKSIZE-1)/(2*BLOCKSIZE)) && LastLevel==0)
	    Auxiliary[gridDim.x*AuxiliaryIndex+blockIdx.x]=Tmp[2*blockDim.x-1];
	   
	    Tmp[2*blockDim.x-1]=0; 
	} 

	for(int d=1; d<2*blockDim.x; d*=2)
	{
		offset>>=1;
		__syncthreads();


		if(tid<d)
		{
		   int ai = offset*(2*tid+1)-1;	
		   int bi = offset*(2*tid+2)-1;	
		
		   int t = Tmp[ai];
		   Tmp[ai]=Tmp[bi];
		   Tmp[bi]+=t;
		
		}	
	}

	__syncthreads();


	if((OffsetTid+2*tid)<Size)                         
	PreScan[OffsetTid+2*tid] =  Tmp[2*tid];

	if((OffsetTid+2*tid+1)<Size)
	PreScan[OffsetTid+2*tid+1] = Tmp[2*tid+1]; 

	__syncthreads();
 
	AuxiliaryIndex++;
	}
}


__host__ void PreScanRecursive(unsigned int** ListAux, unsigned int* ListAuxSize, int CountAuxiliary, int Depth)
{
	if(CountAuxiliary==0) return;
	int LastLevel=0;

	if(ListAuxSize[Depth]<=2*BLOCKSIZE)     
	{
		LastLevel=1;
		KernelPrescanRecursive<<<512, BLOCKSIZE, 2*BLOCKSIZE*sizeof(unsigned int)>>>(ListAux[Depth], NULL, ListAuxSize[Depth], LastLevel);	
	}

	else
	{
		KernelPrescanRecursive<<<512, BLOCKSIZE, 2*BLOCKSIZE*sizeof(unsigned int)>>>(ListAux[Depth], ListAux[Depth+1], ListAuxSize[Depth], LastLevel);	
	}

	PreScanRecursive(ListAux, ListAuxSize, CountAuxiliary-1, Depth+1);

	if(LastLevel==0)
		FixUpScan<<<512,BLOCKSIZE>>>(ListAux[Depth], ListAux[Depth+1], ListAuxSize[Depth]);
	
}

int main()
{
	int Size;
	
	fread(&Size, sizeof(int), 1, stdin);
	
	unsigned int*  Data = (unsigned int*)malloc(Size*sizeof(unsigned int));
	fread(Data, Size*sizeof(unsigned int), 1, stdin);
     
	unsigned int m = getMax(Data, Size);   

	unsigned int*  PreScan;        
	unsigned char*  Bits;          
	unsigned int* Dev_Data;        
	unsigned int* Diff_N_Sn;       
	unsigned int* OutData;
	


	hipMalloc((void**)&Dev_Data, Size*sizeof(unsigned int));
	hipMalloc((void**)&PreScan, Size*sizeof(unsigned int));
	hipMalloc((void**)&Bits, Size*sizeof(unsigned char));
	hipMalloc((void**)&Diff_N_Sn, sizeof(unsigned int));
	hipMalloc((void**)&OutData, Size*sizeof(unsigned int));
	hipMemcpy(Dev_Data, Data, Size*sizeof(unsigned int), hipMemcpyHostToDevice);


	int CountAuxiliary = 0;    
	int PrevAuxSize = Size;
	int NextAuxSize=0;

    do 
	{
		NextAuxSize = (PrevAuxSize+2*BLOCKSIZE-1)/(2*BLOCKSIZE);  
		PrevAuxSize = NextAuxSize;
		CountAuxiliary++;
	} while(NextAuxSize >= 2*BLOCKSIZE);


	unsigned int* ListAuxSize = (unsigned int*)malloc(CountAuxiliary*sizeof(unsigned int));
	
	unsigned int** ListAux;

	ListAux = (unsigned int**)malloc(CountAuxiliary*sizeof(unsigned int*));

	 PrevAuxSize = Size;
	 NextAuxSize=0;

	for(int i=0; i<CountAuxiliary; i++)
	{
	  NextAuxSize = (PrevAuxSize+2*BLOCKSIZE-1)/(2*BLOCKSIZE);
	  ListAuxSize[i] = NextAuxSize;
	  hipMalloc((void**)&ListAux[i], NextAuxSize*sizeof(unsigned int));
	  //cudaMemcpy(ListAux[i], ListSupport[i], NextAuxSize*sizeof(unsigned int), cudaMemcpyHostToDevice);
	  PrevAuxSize = NextAuxSize;
	}

	for (unsigned int bit = 0; (m>>bit) > 0; bit++)
	{
		KernelPrescan<<<512,BLOCKSIZE, 2*BLOCKSIZE*sizeof(unsigned int)>>>(Dev_Data, Bits, PreScan, ListAux[0], Size, bit);   //invoke CountSort by every bit
		PreScanRecursive(ListAux, ListAuxSize, CountAuxiliary, 0);
		FixUpScan<<<512,BLOCKSIZE>>>(PreScan, ListAux[0], Size);
		GetDiff_N_Sn<<<1,1>>>(Bits, PreScan, Diff_N_Sn, Size);
		Sort<<<512,BLOCKSIZE>>>(Dev_Data, PreScan, OutData, Bits, Diff_N_Sn, Size);
		Exchange<<<512,BLOCKSIZE>>>(Dev_Data, OutData, Size);

		if ((m>>bit)==1)  
		{
			break;
		}
	}


	hipMemcpy(Data, Dev_Data, Size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	 
	fwrite(Data, Size*sizeof(unsigned int), 1, stdout);

    return 0;
}
